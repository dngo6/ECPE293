#include "hip/hip_runtime.h"
/*
ssh dduyngo@node009
nvcc -lm -o -level1 convolve.cu 
*/
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "image_template.h"
#include <math.h>
#include<sys/time.h>

//Prints a floating point matrix of given dimensions for logic debugging
void print_matrix(float *image,int width,int height)
{
  int i,j;

  for(i=0;i<height;i++)
  {
    printf("\n");
    for(j=0;j<width;j++)
      printf(" %f",image[i*width+j]); 
  } 
}

//A consolidated function that creates both the Gaussian kernel and the derivative kernel
void create_gaussians(float **gaussian_kernel,float **gaussian_deriv,int k_width,float sigma)
{
  int i,j;
  float sum=0;
  int a=k_width/2;
  printf("\n Creating kernels of width:%d and sigma:%f",k_width,sigma);

  *gaussian_kernel=(float *)malloc(sizeof(float)*k_width);   
  *gaussian_deriv=(float *)malloc(sizeof(float)*k_width);  

  //Create kernel
  sum=0;
  for(i=0;i<k_width;i++)
  {
    (*gaussian_kernel)[i]=exp((-1*(i-a)*(i-a))/(2*sigma*sigma));  
    sum+=(*gaussian_kernel)[i];
  }

  for(i=0;i<k_width;i++)
    (*gaussian_kernel)[i]/=sum;


  //Create derivative
  sum=0;
  for(i=0;i<k_width;i++)
  {
    (*gaussian_deriv)[i]=-1*(i-a)*exp((-1*(i-a)*(i-a))/(2*sigma*sigma));  
    sum-=i*((*gaussian_deriv)[i]);
  }

  for(i=0;i<k_width;i++)
    (*gaussian_deriv)[i]/=sum;

}

//A GPU kernel  for convolution. Input image can be either int or float BUT the output is always float
__global__
void convolve(float *in_image,int width,int height,float *mask,int mask_width,int mask_height,float *out_image)
{
  int i,j,k,m;
  float sum;
  int offseti,offsetj;
 
  i=blockIdx.x*blockDim.x + threadIdx.x;
  j=blockIdx.y*blockDim.y + threadIdx.y;

   if(i<height && j <width)
    {
       sum=0;
       for(k=0;k<mask_height;k++)
       {
         for(m=0;m<mask_width;m++)
         {
           offseti= -1*(mask_height/2)+k;
	   offsetj= -1*(mask_width/2)+m;
           if(i+offseti >=0 && i+offseti<height && j+offsetj>=0 && j+offsetj<width)
           {
              sum+=(float)(in_image[(i+offseti)*width+(j+offsetj)])*mask[k*mask_width+m];
           }                  
         }
       }
       
       out_image[i*width+j]=(float)sum; 
    }

}

int main(int argc, char **argv)
{

  //Declare all of the variable here
  float  *org_img;
  
 //GPU device buffer for original image
 float *d_org_img;

 //CPU host buffers for the final output 
 float  *vertical_gradient,*horizontal_gradient;

 //GPU buffers for the final result
 float *d_vertical_gradient,*d_horizontal_gradient;

  //GPU buffers to hold intermediate convolution results
  float *d_temp_horizontal,*d_temp_vertical;

  //CPU host buffers to store the convolution masks
  float *gaussian_kernel,*gaussian_deriv;

  //GPU device buffers to store the convolution masks
  float *d_gaussian_kernel,*d_gaussian_deriv;

  int width,height,k_width;
  float sigma,a;
  struct timeval start,end;
  if(argc!=3)
  {
    printf("\n The correct argument list is: exec <image file> <Sigma> \n");
    exit(0);
  }
 
  //obtain the parameters
  sigma=atof(argv[2]);
  a=ceil((float)(2.5*sigma-0.5));
  k_width=2*a+1;
 
  //CPU portion of the code that reads/prepares the input data
  read_image_template<float>(argv[1],&org_img,&width,&height);    
 
  //Computation starts here
  gettimeofday(&start,NULL);

  create_gaussians(&gaussian_kernel,&gaussian_deriv,k_width,sigma);
  
  //Allocate for intermediate images
//  temp_horizontal=(float *)malloc(sizeof(float)*width*height);
//  temp_vertical=(float *)malloc(sizeof(float)*width*height);

  horizontal_gradient=(float *)malloc(sizeof(float)*width*height);
  vertical_gradient=(float *)malloc(sizeof(float)*width*height);

// CPU host mallocs for GPU buffers
 hipMalloc((void **)&d_org_img,sizeof(float)*width*height);
 hipMalloc((void **)&d_temp_horizontal,sizeof(float)*width*height);
 hipMalloc((void **)&d_temp_vertical,sizeof(float)*width*height);
 hipMalloc((void **)&d_horizontal_gradient,sizeof(float)*width*height);
 hipMalloc((void **)&d_vertical_gradient,sizeof(float)*width*height);

 hipMalloc((void **)&d_gaussian_kernel,sizeof(float)*k_width);
 hipMalloc((void **)&d_gaussian_deriv,sizeof(float)*k_width);

  //Check kernels
  
  printf("\n The gaussian kernel is:");
  print_matrix(gaussian_kernel,k_width,1);
  
  printf("\n The gaussian derivative is:");
  print_matrix(gaussian_deriv,k_width,1);

  printf("\n");

//Offload all of the data to GPU device for convolution
hipMemcpy(d_org_img,org_img,sizeof(float)*width*height,hipMemcpyHostToDevice);

hipMemcpy(d_gaussian_kernel,gaussian_kernel,sizeof(float)*k_width,hipMemcpyHostToDevice);
hipMemcpy(d_gaussian_deriv,gaussian_deriv,sizeof(float)*k_width,hipMemcpyHostToDevice);


  //Horizontal gradient. vertical kernel then horizontal derivative
int block_dim=16;
 dim3 dimGrid(ceil(height/block_dim),ceil(width/block_dim),1);
 dim3 dimBlock(block_dim,block_dim,1);

  convolve<<<dimGrid,dimBlock>>>(d_org_img,width,height,d_gaussian_kernel,1,k_width,d_temp_horizontal); 
  convolve<<<dimGrid,dimBlock>>>(d_temp_horizontal,width,height,d_gaussian_deriv,k_width,1,d_horizontal_gradient);

  //Vertical gradient. horizontal kernel then vertical derivative
  convolve<<<dimGrid,dimBlock>>>(d_org_img,width,height,d_gaussian_kernel,k_width,1,d_temp_vertical); 
  convolve<<<dimGrid,dimBlock>>>(d_temp_vertical,width,height,d_gaussian_deriv,1,k_width,d_vertical_gradient);
  
//GPU to Host transfer of the final result

hipMemcpy(horizontal_gradient,d_horizontal_gradient,sizeof(float)*width*height,hipMemcpyDeviceToHost);
hipMemcpy(vertical_gradient,d_vertical_gradient,sizeof(float)*width*height,hipMemcpyDeviceToHost);

hipDeviceSynchronize();

  gettimeofday(&end,NULL);

  printf("Execution time in ms: %ld\n", ((end.tv_sec * 1000 + end.tv_usec/1000)
		  - (start.tv_sec * 1000 + start.tv_usec/1000)));

  write_image_template<float>((char *)("horizontal_gradient.pgm"),horizontal_gradient,width,height);
  write_image_template<float>((char *)("vertical_gradient.pgm"),vertical_gradient,width,height);
 
  //free variables
  free(org_img);
//  free(temp_horizontal);
//  free(temp_vertical);
  free(horizontal_gradient);
  free(vertical_gradient);
  free(gaussian_kernel);
  free(gaussian_deriv);

  hipFree(d_org_img);
  hipFree(d_gaussian_kernel);
  hipFree(d_gaussian_deriv);
  hipFree(d_temp_horizontal);
  hipFree(d_temp_vertical);
  hipFree(d_vertical_gradient);
  hipFree(d_horizontal_gradient);
  return 0;
}
